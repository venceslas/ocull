#include "hip/hip_runtime.h"

#include "PassThrough.hpp"
#include "../cuda/PixelPipe.inl"


using namespace FW;


//==============================================================================

//------------------------------------------------------------------------
// Vertex shaders.
//------------------------------------------------------------------------

// test of a simple 'passthrough' vertex shader
extern "C" __global__
void FW::vertexShader_passthrough( const InputVertex* inPtr,              // IN
                                   ShadedVertex_passthrough* outPtr,      // OUT
                                   int numVertices)                       // IN
{
  int VertexID = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * 
                 (blockIdx.x + gridDim.x * blockIdx.y));

  if (VertexID >= numVertices) {
    return;
  }
  
  const InputVertex&        in  = inPtr[VertexID];
  ShadedVertex_passthrough& out = outPtr[VertexID];
  
  Vec4f inPosition      = Vec4f( in.modelPos, 1.0f);
  Mat4f &uModelViewProj = c_constants.posToClip;
  
  out.clipPos = uModelViewProj * inPosition;
}


//==============================================================================

//------------------------------------------------------------------------
// Fragment shaders.
//------------------------------------------------------------------------

class FragmentShader_passthrough : public FragmentShaderBase
{
  public:
    // Override
    __device__ __inline__ void run(void)
    {
      Vec3f red_color = Vec3f( 1.0f, 0.0f, 0.0f);
      
      m_color = toABGR( Vec4f( red_color, 1.0f) );
    }
};


//==============================================================================

//------------------------------------------------------------------------
// Pixel pipes.
//------------------------------------------------------------------------


CR_DEFINE_PIXEL_PIPE( PixelPipe_passthrough,
                      ShadedVertex_passthrough, 
                      FragmentShader_passthrough, BLEND_SHADER, 
                      SAMPLES_LOG2, RENDER_MODE_FLAGS)


//==============================================================================

